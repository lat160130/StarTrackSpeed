#include "hip/hip_runtime.h"
// This program computes matrix multiplication using shared memory tiling
// By: Nick from CoffeeBeforeArch

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include "fstream" 
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include "math_func.h"
#include "help_func.h"     
#include "string"
#include <cmath>   
#define NUMDIMS 3

using namespace std;

using std::cout;
using std::generate;
using std::vector;

// Pull out matrix and shared memory tile size
const int M = 1 << 10;
const int N = 1 << 11;
const int K = 1 << 12;
const int SHMEM_SIZE = 1 << 10;

__global__ void matrixMul(const double *a, const double *b, double *c) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Statically allocated shared memory
  __shared__ int s_a[SHMEM_SIZE];
  __shared__ int s_b[SHMEM_SIZE];

  // Accumulate in temporary variable
  int tmp = 0;

  // Sweep tile across matrix
  for (int i = 0; i < K; i += blockDim.x) {
    // Load in elements for this tile
    s_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * K + i + threadIdx.x];
    s_b[threadIdx.y * blockDim.x + threadIdx.x] =
        b[i * N + threadIdx.y * N + col];

    // Wait for both tiles to be loaded in before doing computation
    __syncthreads();

    // Do matrix multiplication on the small matrix
    for (int j = 0; j < blockDim.x; j++) {
      tmp +=
          s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
    }

    // Wait for all threads to finish using current tiles before loading in new
    // ones
    __syncthreads();
  }

  // Write back results
  c[row * N + col] = tmp;
}

// Check result on the CPU
// MxN = MxK * KxN
void verify_result(vector<double> &a, vector<double> &b, vector<double> &c) {
  // For every row...
  for (int row = 0; row < M; row++) {
    // For every column...
    for (int col = 0; col < N; col++) {
      // For every element in the row-column pair
      double tmp = 0;
      for (int i = 0; i < K; i++) {
        // Accumulate the partial results
        tmp += a[row * K + i] * b[i * N + col];
      }

      // Check against the CPU result
      assert(tmp == c[row * N + col]);
    }
  }
}

int main() {
  // Size (in bytes) of matrix
  // MxN = MxK * KxN
  size_t bytes_a = M * K * sizeof(double);
  size_t bytes_b = K * N * sizeof(double);
  size_t bytes_c = M * N * sizeof(double);

  // Host vectors
  vector<double> h_a(M * K);
  vector<double> h_b(K * N);
  vector<double> h_c(M * N);


int rows = 128;
const char txtMatObs[] = "vectorInObsCM.txt";
const char txtMatRef[] = "vectorInRef.txt";


ifstream fpMatObs(txtMatObs);
ifstream fpMatRef(txtMatRef);
// Check if either text file failed to open
if ((!fpMatObs) || (!fpMatRef)){
    perror("Text file opening failed: vectorInObs.txt or vectorInRef.txt failed to open.");
    return 1;
} // end if
double *matObs = (double*) malloc(rows*NUMDIMS * sizeof(double));
double *matRef = (double*) malloc(rows*NUMDIMS * sizeof(double));

cout << "readin data" << endl;
for (int i = 0; i < rows*NUMDIMS; i++){

    fpMatObs >> h_a[i];//matObs[i];
    fpMatRef >> h_b[i];//matRef[i];
} // end for x
cout << "read data" << endl;

cout << "verify data" << endl;
for (int i = 0; i < rows*NUMDIMS; i++){

    cout << h_a[i];//matObs[i];
    // fpMatRef >> h_b[i];//matRef[i];
} // end for x


  // Allocate device memory
  double *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes_a);
  hipMalloc(&d_b, bytes_b);
  hipMalloc(&d_c, bytes_c);

  // Copy data to the device
  hipMemcpy(d_a, h_a.data(), bytes_a, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes_b, hipMemcpyHostToDevice);

  // Threads per CTA dimension
  int THREADS = 32;

  // Blocks per grid dimension (assumes THREADS divides M and N evenly)
  int BLOCKS_X = rows / THREADS;
  int BLOCKS_Y = M / THREADS;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS_X, 1);

  // Launch kernel
  matrixMul<<<blocks, threads>>>(d_a, d_b, d_c);

  // Copy back to the host
  hipMemcpy(h_c.data(), d_c, bytes_c, hipMemcpyDeviceToHost);
 for (int i = 0; i < NUMDIMS * NUMDIMS; i ++){
    cout << "h_c[" << i << "] = " << h_c[i] << endl;
  } // for

  // Check result
  verify_result(h_a, h_b, h_c);

 

  cout << "COMPLETED SUCCESSFULLY\n";

  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}